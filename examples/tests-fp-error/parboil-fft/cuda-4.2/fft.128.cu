#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <parboil.h>
#include <assert.h>
#include "gqd.cu"
#include "gqd_type.h"
#include <qd/qd_real.h>

#include <iostream>
using namespace std;


#ifndef NN
#define NN 4
#endif 
#ifndef BB
#define BB 4 
#endif 

#ifndef GPUID
#define GPUID 0
#endif 

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

// Block index
#define  bx  blockIdx.x
#define  by  blockIdx.y
// Thread index
#define tx  threadIdx.x

// Possible values are 2, 4, 8 and 16
#ifndef R 
#define R 2
#endif 

typedef struct {
  gdd_real x;
  gdd_real y;
} gdd_real2; 

inline gdd_real2 __device__ make_gdd_real2 (gdd_real inx, gdd_real iny) {
  gdd_real2 ret;
  ret.x = inx; ret.y = iny;
  return ret;
}

inline gdd_real2 __device__ operator*( gdd_real2 a, gdd_real2 b ) { return make_gdd_real2( a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x ); }
inline gdd_real2 __device__ operator+( gdd_real2 a, gdd_real2 b ) { return make_gdd_real2( a.x + b.x, a.y + b.y ); }
inline gdd_real2 __device__ operator-( gdd_real2 a, gdd_real2 b ) { return make_gdd_real2( a.x - b.x, a.y - b.y ); }
inline gdd_real2 __device__ operator*( gdd_real2 a, gdd_real b ) { return make_gdd_real2( b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_gdd_real2(  make_dd((double)COS_PI_8), make_dd((double)-SIN_PI_8) )
#define exp_3_16  make_gdd_real2(  make_dd((double)SIN_PI_8), make_dd((double)-COS_PI_8) )
#define exp_5_16  make_gdd_real2( make_dd((double)-SIN_PI_8), make_dd((double)-COS_PI_8) )
#define exp_7_16  make_gdd_real2( make_dd((double)-COS_PI_8), make_dd((double)-SIN_PI_8) )
#define exp_9_16  make_gdd_real2( make_dd((double)-COS_PI_8),  make_dd((double)SIN_PI_8) )
#define exp_1_8   make_gdd_real2(  make_dd(1.0), make_dd(-1.0) )
#define exp_1_4   make_gdd_real2(  make_dd(0.0), make_dd(-1.0) )
#define exp_3_8   make_gdd_real2( make_dd(-1.0), make_dd(-1.0) )

dd_real gdd_to_dd (gdd_real fd) {
  dd_real ret;
  ret.x[0] = fd.x;
  ret.x[1] = fd.y;
  return ret;
}

void inputData(FILE* infile, gdd_real2* dat, int num_gdd_real2)
{
  for (unsigned int i = 0 ; i < num_gdd_real2 ; i++) {
    fread(&(dat[i].x.x), sizeof(double), 1, infile);
    fread(&(dat[i].x.y), sizeof(double), 1, infile);
    fread(&(dat[i].y.x), sizeof(double), 1, infile);
    fread(&(dat[i].y.y), sizeof(double), 1, infile);
  }
}

void outputData(FILE* outfile, gdd_real outdat)
{
  cout << "out_data : " << gdd_to_dd(outdat) << endl;
  fwrite(&(outdat.x), sizeof(double), 1, outfile);
  fwrite(&(outdat.y), sizeof(double), 1, outfile);
}
  
__device__ void GPU_FFT2( gdd_real2 &v1,gdd_real2 &v2 ) { 
  gdd_real2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( gdd_real2 &v0,gdd_real2 &v1,gdd_real2 &v2,gdd_real2 &v3) { 
   GPU_FFT2(v0, v2);
   GPU_FFT2(v1, v3);
   v3 = v3 * exp_1_4;
   GPU_FFT2(v0, v1);
   GPU_FFT2(v2, v3);    
}


inline __device__ void GPU_FFT2(gdd_real2* v){
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(gdd_real2* v){
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}


inline __device__ void GPU_FFT8(gdd_real2* v){
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);

  v[5]=(v[5]*exp_1_8)*make_dd(M_SQRT1_2);
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*make_dd(M_SQRT1_2);

  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
  
}

inline __device__ void GPU_FFT16( gdd_real2 *v )
{
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * make_dd(M_SQRT1_2);
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * make_dd(M_SQRT1_2);
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * make_dd(M_SQRT1_2);
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * make_dd(M_SQRT1_2);
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}
     
__device__ int GPU_expand(int idxL, int N1, int N2 ){ 
  return (idxL/N1)*N1*N2 + (idxL%N1); 
}      

__device__ void GPU_FftIteration(int j, int Ns, gdd_real2* data0, gdd_real2* data1){ 
  gdd_real2 v[R];  	
  int idxS = j;       
  gdd_real angle = make_dd((-2*M_PI*(j%Ns)/(Ns*R)));

  for( int r=0; r<R; r++ ) { 
    v[r] = data0[idxS+r*NN/R]; 
    double r64 = (double)r;
    double angle64 = to_double(angle);
    double cos64 = cos(r64 * angle64); // call cuda "cos" instead of the function provided by QD_REAL library 
    double sin64 = sin(r64 * angle64); // call cuda "sin" instead of the function provided by QD_REAL library 
    gdd_real cos_gdd = make_dd(cos64);
    gdd_real sin_gdd = make_dd(sin64);

    v[r] = v[r] * make_gdd_real2(cos_gdd, sin_gdd);
  }       

#if R == 2 
  GPU_FFT2( v ); 
#endif

#if R == 4
  GPU_FFT4( v );
#endif	 	

#if R == 8
  GPU_FFT8( v );
#endif

#if R == 16
  GPU_FFT16( v );
#endif	 	

  int idxD = GPU_expand(j,Ns,R); 

  for( int r=0; r<R; r++ ){
    data1[idxD+r*Ns] = v[r];	
  } 	

}      

__global__ void GPU_FFT_Global(int Ns, gdd_real2* data0, gdd_real2* data1) { 
  data0+=bx*NN;
  data1+=bx*NN;	 
  GPU_FftIteration( tx, Ns, data0, data1);  
}      

int main( int argc, char **argv )
{	
  int n_bytes; 

  // check config 
  assert(NN > 0);
  assert(BB > 0);
  assert(R > 0);
  unsigned int nn = NN;
  unsigned int rr = R; 
  while (nn > 0) {
    if (nn == 1) break;
    assert(nn / rr > 0); 
    nn = nn / rr; 
  }    
  
  // get IO file name 
  assert(argc == 3);
  char *inname = argv[1];
  char *outname = argv[2];
  
  // int N, B;
  n_bytes = NN*BB*sizeof(gdd_real2);

  hipSetDevice(GPUID);

  gdd_real2 *source;
  gdd_real2 *result;
  hipHostMalloc((void**)&source, n_bytes);
  CUERR;
  hipHostMalloc((void**)&result, n_bytes);
  CUERR;

  // allocate device memory
  gdd_real2 *d_source, *d_work;
  hipMalloc((void**) &d_source, n_bytes);
  CUERR;
  hipMalloc((void**) &d_work, n_bytes);
  CUERR;

  FILE * infile = fopen(inname, "r");
  assert(infile != NULL);
  fseek(infile, 0, SEEK_END);
  long fsize = ftell(infile);
  fseek(infile, 0, SEEK_SET);
  assert(fsize % (NN*BB*sizeof(double)*4) == 0);
  unsigned int n_repeats = fsize / (NN*BB*sizeof(double)*4);

  FILE *outfile = fopen(outname, "w");
  assert(outfile != NULL);

  for (unsigned int ri = 0 ; ri < n_repeats ; ri++) {

    inputData(infile, (gdd_real2*)source,NN*BB);
  
    // copy host memory to device
    hipMemcpy(d_source, source, n_bytes,hipMemcpyHostToDevice);
    CUERR;
    hipMemset(d_work, 0,n_bytes);
    CUERR;

    for( int Ns=1; Ns<NN; Ns*=R){
      GPU_FFT_Global<<<dim3(BB), dim3(NN/R)>>>(Ns, d_source, d_work);
      gdd_real2 *tmp = d_source;
      d_source = d_work;
      d_work = tmp;
    }

    // copy device memory to host
    hipMemcpy(result, d_source, n_bytes,hipMemcpyDeviceToHost);
    CUERR;

    outputData(outfile, (gdd_real)result[NN*BB-1].y);

  }

  hipFree(d_source);
  CUERR;
  hipFree(d_work);
  CUERR;	

  hipHostFree(source);
  hipHostFree(result);

  fclose(infile);
  fclose(outfile);

  return 0;
}

