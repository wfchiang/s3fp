#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

#include "dataio.h" 

using namespace std;


#ifndef FT 
#define FT float 
#define FT2 float2
#define make_FT2 make_float2
#endif

#ifndef IFT
#define IFT float 
#endif  

#ifndef NN  
#define NN 4
#endif 
#ifndef BB
#define BB 4 
#endif 

#ifndef GPUID
#define GPUID 0
#endif 

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

// Block index
#define  bx  blockIdx.x
#define  by  blockIdx.y
// Thread index
#define tx  threadIdx.x

// Possible values are 2, 4, 8 and 16
#ifndef R 
#define R 2
#endif 

inline FT2 __device__ operator*( FT2 a, FT2 b ) { return make_FT2( a.x*b.x-a.y*b.y, a.x*b.y+a.y*b.x ); }
inline FT2 __device__ operator+( FT2 a, FT2 b ) { return make_FT2( a.x + b.x, a.y + b.y ); }
inline FT2 __device__ operator-( FT2 a, FT2 b ) { return make_FT2( a.x - b.x, a.y - b.y ); }
inline FT2 __device__ operator*( FT2 a, FT b ) { return make_FT2( b*a.x , b*a.y); }

#define COS_PI_8  0.923879533f
#define SIN_PI_8  0.382683432f
#define exp_1_16  make_FT2(  COS_PI_8, -SIN_PI_8 )
#define exp_3_16  make_FT2(  SIN_PI_8, -COS_PI_8 )
#define exp_5_16  make_FT2( -SIN_PI_8, -COS_PI_8 )
#define exp_7_16  make_FT2( -COS_PI_8, -SIN_PI_8 )
#define exp_9_16  make_FT2( -COS_PI_8,  SIN_PI_8 )
#define exp_1_8   make_FT2(  1, -1 )
#define exp_1_4   make_FT2(  0, -1 )
#define exp_3_8   make_FT2( -1, -1 )

void inputData(FILE* fid, FT2* dat, int num_ft2)
{
  assert(sizeof(FT) == 4 || sizeof(FT) == 8); 

  if (fid == NULL)
    {
      fprintf(stderr, "Cannot open input file\n");
      exit(-1);
    }
  for (unsigned int i = 0 ; i < num_ft2 ; i++) {
    IFT in_data;
    fread(&(in_data), sizeof(IFT), 1, fid);
    if (sizeof(FT) == 4) 
      dat[i].x = (float) in_data;
    else // (sizeof(FT) == 8) 
      dat[i].x = (double) in_data; 
    fread(&(in_data), sizeof(IFT), 1, fid);
    if (sizeof(FT) == 4) 
      dat[i].y = (float) in_data; 
    else // (sizeof(FT) == 8) 
      dat[i].y = (double) in_data; 
  }
}

  
__device__ void GPU_FFT2( FT2 &v1,FT2 &v2 ) { 
  FT2 v0 = v1;  
  v1 = v0 + v2; 
  v2 = v0 - v2; 
}

__device__ void GPU_FFT4( FT2 &v0,FT2 &v1,FT2 &v2,FT2 &v3) { 
   GPU_FFT2(v0, v2);
   GPU_FFT2(v1, v3);
   v3 = v3 * exp_1_4;
   GPU_FFT2(v0, v1);
   GPU_FFT2(v2, v3);    
}


inline __device__ void GPU_FFT2(FT2* v){
  GPU_FFT2(v[0],v[1]);
}

inline __device__ void GPU_FFT4(FT2* v){
  GPU_FFT4(v[0],v[1],v[2],v[3] );
}


inline __device__ void GPU_FFT8(FT2* v){
  GPU_FFT2(v[0],v[4]);
  GPU_FFT2(v[1],v[5]);
  GPU_FFT2(v[2],v[6]);
  GPU_FFT2(v[3],v[7]);

  v[5]=(v[5]*exp_1_8)*M_SQRT1_2;
  v[6]=v[6]*exp_1_4;
  v[7]=(v[7]*exp_3_8)*M_SQRT1_2;

  GPU_FFT4(v[0],v[1],v[2],v[3]);
  GPU_FFT4(v[4],v[5],v[6],v[7]);
  
}

inline __device__ void GPU_FFT16( FT2 *v )
{
    GPU_FFT4( v[0], v[4], v[8], v[12] );
    GPU_FFT4( v[1], v[5], v[9], v[13] );
    GPU_FFT4( v[2], v[6], v[10], v[14] );
    GPU_FFT4( v[3], v[7], v[11], v[15] );

    v[5]  = (v[5]  * exp_1_8 ) * M_SQRT1_2;
    v[6]  =  v[6]  * exp_1_4;
    v[7]  = (v[7]  * exp_3_8 ) * M_SQRT1_2;
    v[9]  =  v[9]  * exp_1_16;
    v[10] = (v[10] * exp_1_8 ) * M_SQRT1_2;
    v[11] =  v[11] * exp_3_16;
    v[13] =  v[13] * exp_3_16;
    v[14] = (v[14] * exp_3_8 ) * M_SQRT1_2;
    v[15] =  v[15] * exp_9_16;

    GPU_FFT4( v[0],  v[1],  v[2],  v[3] );
    GPU_FFT4( v[4],  v[5],  v[6],  v[7] );
    GPU_FFT4( v[8],  v[9],  v[10], v[11] );
    GPU_FFT4( v[12], v[13], v[14], v[15] );
}
     
__device__ int GPU_expand(int idxL, int N1, int N2 ){ 
  return (idxL/N1)*N1*N2 + (idxL%N1); 
}      

__device__ void GPU_FftIteration(int j, int Ns, FT2* data0, FT2* data1){ 
  FT2 v[R];  	
  int idxS = j;       
  FT angle = -2*M_PI*(j%Ns)/(Ns*R);      

  for( int r=0; r<R; r++ ) { 
    v[r] = data0[idxS+r*NN/R]; 
    v[r] = v[r]*make_FT2(cos(r*angle), sin(r*angle)); 
  }       

#if R == 2 
  GPU_FFT2( v ); 
#endif

#if R == 4
  GPU_FFT4( v );
#endif	 	

#if R == 8
  GPU_FFT8( v );
#endif

#if R == 16
  GPU_FFT16( v );
#endif	 	

  int idxD = GPU_expand(j,Ns,R); 

  for( int r=0; r<R; r++ ){
    data1[idxD+r*Ns] = v[r];	
  } 	

}      

__global__ void GPU_FFT_Global(int Ns, FT2* data0, FT2* data1) { 
  data0+=bx*NN;
  data1+=bx*NN;	 
  GPU_FftIteration( tx, Ns, data0, data1);  
}      

int main( int argc, char **argv )
{	
  int n_bytes; 

  // check config 
  assert(NN > 0);
  assert(BB > 0);
  assert(R > 0);
  unsigned int nn = NN;
  unsigned int rr = R; 
  while (nn > 0) {
    if (nn == 1) break;
    assert(nn / rr > 0); 
    nn = nn / rr; 
  }    

  // get IO file names
  assert(argc == 3);
  char *inname = argv[1]; 
  char *outname = argv[2];

  // int N, B;
  n_bytes = NN*BB*sizeof(FT2);

  hipSetDevice(GPUID);
  
  FT2 *source;
  FT2 *result;
  hipHostMalloc((void**)&source, n_bytes);
  CUERR;
  hipHostMalloc((void**)&result, n_bytes);
  CUERR;

  FT2 *d_source, *d_work;
  hipMalloc((void**) &d_source, n_bytes);
  CUERR;
  hipMalloc((void**) &d_work, n_bytes);
  CUERR;

  FILE * infile = fopen(inname, "r");
  assert(infile != NULL);
  fseek(infile, 0, SEEK_END);
  long fsize = ftell(infile);
  fseek(infile, 0, SEEK_SET);
  assert(fsize % (NN*BB*sizeof(IFT)*2) == 0);
  unsigned int n_repeats = fsize / (NN*BB*sizeof(IFT)*2);

  FILE *outfile = fopen(outname, "w");
  assert(outfile != NULL);

  for (unsigned int ri = 0 ; ri < n_repeats ; ri++) {

    inputData(infile,(FT2*)source,NN*BB);

    // copy host memory to device
    hipMemcpy(d_source, source, n_bytes,hipMemcpyHostToDevice);
    CUERR;
    hipMemset(d_work, 0,n_bytes);
    CUERR;
    
    for( int Ns=1; Ns<NN; Ns*=R){
      GPU_FFT_Global<<<dim3(BB), dim3(NN/R)>>>(Ns, d_source, d_work);
      FT2 *tmp = d_source;
      d_source = d_work;
      d_work = tmp;
    }

    hipMemcpy(result, d_source, n_bytes,hipMemcpyDeviceToHost);
    CUERR;

    writeOutput64(outfile, (FT)result[NN*BB-1].y);
  }

  hipFree(d_source);
  CUERR;
  hipFree(d_work);
  CUERR;	

  hipHostFree(source);
  hipHostFree(result);

  fclose(infile);
  fclose(outfile);

  return 0;
}

